#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#pragma comment (lib, "hiprand.lib")

#define MY_MAX_INT 9
#define MY_MIN_INT 1

void Print(unsigned int* a, const size_t nrow, const size_t ncol)
{
    for (size_t i = 0; i < nrow; i++)
    {
        for (size_t j = 0; j < ncol; j++)
        {
            std::cout << a[i * ncol + j] << " ";
        }
        std::cout << std::endl;
    }
}




#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

__global__ void scaleToMinMax(unsigned int* a, const size_t size, const int min, const int max) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        *(a+i)%= (max - min) + min+1;
    } 
}

unsigned int main()
{
    unsigned int* dev_a = 0;
    unsigned int* dev_b = 0;
    unsigned int* host_a;
    unsigned int* host_b;

    constexpr size_t NRowBlock = 2;
    constexpr size_t NElementsInBlock = 4;
    constexpr size_t NRowElements = NRowBlock*NElementsInBlock;
    constexpr size_t NCol = 2;
    constexpr size_t dataSize = NRowElements * NCol;

//    hipError_t cudaStatus;

    hiprandGenerator_t gen;

    /* Allocate n floats on host */
    host_a = (unsigned int*)calloc(dataSize, sizeof(unsigned int));

    /* Allocate n floats on host */
    host_b = (unsigned int*)calloc(dataSize, sizeof(unsigned int));

    /* Allocate n floats on device */
    CUDA_CALL(hipMalloc((void**)&dev_a, dataSize * sizeof(unsigned int)));

    /* Allocate n floats on device */
    CUDA_CALL(hipMalloc((void**)&dev_b, dataSize * sizeof(unsigned int)));

    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen,
        HIPRAND_RNG_PSEUDO_DEFAULT));

    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,
        1234ULL));

    /* Generate n floats on device */
    CURAND_CALL(hiprandGenerate(gen, dev_a, dataSize));

    ////////////////////////////////////////////////////////////////

    scaleToMinMax << <dataSize,1 >> > (dev_a, dataSize, MY_MIN_INT, MY_MAX_INT);


    ////////////////////////////////////////////////////////////////

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(host_a, dev_a, dataSize* sizeof(unsigned int),
        hipMemcpyDeviceToHost));

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(host_b, dev_b, dataSize * sizeof(unsigned int),
        hipMemcpyDeviceToHost));

    // hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
    CUDA_CALL(hipDeviceReset());

    /* Show result */
    Print(host_a, NRowElements, NCol);
    printf("\n");

    Print(host_b, NCol, NRowElements);
    printf("\n");

    /* Cleanup */
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(dev_a));
    CUDA_CALL(hipFree(dev_b));
    free(host_a);
    free(host_b);

    return 0;
}

